#include "hip/hip_runtime.h"
#include <cudaviz/Mandelbrot>
#include <iostream>

namespace cudaviz
{
    namespace device
    {

        __device__ float scale(int k, int N, float min, float max)
        {
            return min + k * ((max - min) / N);
        }

        __global__ void mandelbrot_iteration(int *grid, int N, int max_iter, float xcenter, float y_center, float zoom)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int j = blockIdx.y * blockDim.y + threadIdx.y;
            float scaled_x_width = 2.47 * zoom;
            float scaled_y_height = 2.24 * zoom;
            double x = 0;
            double y = 0;
            double xnew = 0;
            double ynew = 0;

            double x0 = scale(i, N, xcenter - scaled_x_width / 2, xcenter + scaled_x_width / 2);
            double y0 = scale(j, N, y_center - scaled_y_height / 2, y_center + scaled_y_height / 2);

            if (i < N && j < N)
            {
                int index = j * N + i;
                grid[index] = max_iter;
                for (int iter = 0; iter < max_iter; ++iter)
                {
                    xnew = x * x - y * y + x0;
                    ynew = 2 * x * y + y0;
                    if (xnew * xnew + ynew * ynew > 4)
                    {
                        grid[index] = iter;
                    }
                    x = xnew;
                    y = ynew;
                }
            }
        }

        __global__ void naiive_diffusion_iteration(float *d_old, float *d_new, int nx, int ny, float alpha)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;
            int offset = y * nx + x;

            if (x < nx && y < ny)
            {
                int idx_left = offset - 1;
                int idx_right = offset + 1;

                int idx_top = offset - nx;
                int idx_bottom = offset + nx;


                float top, bottom, left, right;
                top = (y > 0) ? d_old[idx_top] : 0;
                bottom = (y < ny - 1) ? d_old[idx_bottom] : 0;
                left = (x > 0) ? d_old[idx_left] : 0;
                right = (x < nx - 1) ? d_old[idx_right] : 0;

                d_new[offset] = d_old[offset] + alpha * (top + bottom + left + right - d_old[offset] * 4.0f);
                if (d_new[offset] < 0.0f)
                {
                    d_new[offset] = 0.0f;
                }
            }
        }
    }

    void mandelbrot_iteration(int *grid, int N, int max_iter, float xcenter, float y_center, float zoom)
    {
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
        device::mandelbrot_iteration<<<numBlocks, threadsPerBlock>>>(grid, N, max_iter, xcenter, y_center, zoom);
        hipDeviceSynchronize();
    }

    void naiive_diffusion_iteration(float *d_old, float *d_new, int nx, int ny, float alpha)
    {
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x, (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);
        device::naiive_diffusion_iteration<<<numBlocks, threadsPerBlock>>>(d_old, d_new, nx, ny, alpha);
        hipDeviceSynchronize();
    }
}