#include "hip/hip_runtime.h"
#include <cudaviz/cudaviz.hpp>

namespace cudaviz
{
    namespace device
    {
        __global__ void setIndex(int *data)
        {
            int idx = threadIdx.x + blockIdx.x * blockDim.x;
            data[idx] = idx;
        }

        __global__ void add(float *A, float *B, float *C)
        {
            int i = threadIdx.x;
            C[i] = A[i] + B[i];
            ;
        }

        __global__ void matAdd(float* A, float* B, float* C, int N) {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int j = blockIdx.y * blockDim.y + threadIdx.y;

            if (i < N && j < N) {
                int index = i*N + j;
                C[index] = A[index] + B[index];
            }
        }
    }

    void setIndex(int *data)
    {
        device::setIndex<<<4, 4>>>(data);
        hipDeviceSynchronize();
    }

    void add(float *A, float *B, float *C, int N)
    {
        device::add<<<1, N>>>(A, B, C);
        hipDeviceSynchronize();
    }

    void matAdd(float* A, float* B, float* C, int N) {
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
        device::matAdd<<<numBlocks, threadsPerBlock>>>(A, B, C, N);
        hipDeviceSynchronize();
    }
}