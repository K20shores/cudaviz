#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

// Kernel function to add the elements of two arrays
__global__
void testKernel(float *x)
{
  x[0] = 123.456f;
}

int main(void)
{
  // Query and print GPU properties
  hipDeviceProp_t prop;
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  std::cout << "Running on GPU: " << prop.name << std::endl;
  std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;

  float *x;
  hipError_t err;

  // Allocate unified memory and check for errors
  err = hipMallocManaged(&x, sizeof(float));
  if (err != hipSuccess) {
    std::cerr << "hipMallocManaged failed: " << hipGetErrorString(err) << std::endl;
    return -1;
  }

  x[0] = 0.0f;

  // Launch kernel and check for errors
  testKernel<<<1, 1>>>(x);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    hipFree(x);
    return -1;
  }

  // Synchronize and check for errors
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(err) << std::endl;
    hipFree(x);
    return -1;
  }

  std::cout << "x[0] = " << x[0] << std::endl;

  // Free memory
  hipFree(x);
  return 0;
}