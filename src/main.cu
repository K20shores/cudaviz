#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <format>

#include "check_error.hpp"

__global__ void add(int a, int b, int* c) {
  *c = a + b;
}

void device_data() {
  int device_count;

  CUDA_CHECK(hipGetDeviceCount(&device_count));

  hipDeviceProp_t prop;

  for(int i = 0; i < device_count; ++i) {
    CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    std::cout << std::format("Device {}\n", i);
  }
}

int main () {
  int c;
  int* dev_c;

  CUDA_CHECK(hipMalloc((void**)&dev_c, sizeof(int)));

  add<<<1, 1>>>(2, 3, dev_c);

  CUDA_CHECK(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

  std::cout << "c: " << c << std::endl;

  CUDA_CHECK(hipFree(dev_c));
  device_data();
}