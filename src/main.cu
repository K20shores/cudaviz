#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <format>

#include "check_error.hpp"
#include <cudaviz/cudaviz>

constexpr int threadsPerBlock = 256;

namespace kernels
{
  __global__ void add(int *a, int *b, int *c, int N)
  {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N)
    {
      c[tid] = a[tid] + b[tid];
      tid += blockDim.x * gridDim.x;
    }
  }

  __global__ void dot(float *a, float *b, float* c, int N)
  {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIdx = threadIdx.x;

    float temp = 0;
    while (tid < N)
    {
      temp += a[tid] * b[tid];
      tid += blockDim.x * gridDim.x;
    }

    cache[cacheIdx] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0)
    {
      if (cacheIdx < i)
      {
        cache[cacheIdx] += cache[cacheIdx + i];
      }
      __syncthreads();
      i /= 2;
    }

    if (cacheIdx == 0) {
      c[blockIdx.x] = cache[0];
    }
  }

  __global__ void histogram(unsigned char* buffer, int N, unsigned int * hist) {
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    __syncthreads();

    while (i < N) {
      atomicAdd(&(temp[buffer[i]]), 1);
      i += stride;
    }
    __syncthreads();

    atomicAdd(&(hist[threadIdx.x]), temp[threadIdx.x]);
  }

  __global__ void average(int *a, int *b, int *c, int N)
  {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
      int idx1 = (idx + 1) % 256;
      int idx2 = (idx + 2) % 256;
      float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
      float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
      c[idx] = (as + bs) / 2.0f;
    }
  }
}

void device_data()
{
  int device_count;

  CUDA_CHECK(hipGetDeviceCount(&device_count));

  hipDeviceProp_t prop;

  for (int i = 0; i < device_count; ++i)
  {
    CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    std::cout << std::format(
        "Device {}\n"
        "  Name: {}\n"
        "  Total Global Memory: {} bytes\n"
        "  Shared Memory per Block: {} bytes\n"
        "  Registers per Block: {}\n"
        "  Warp Size: {}\n"
        "  Max Threads per Block: {}\n"
        "  Max Threads Dim: ({}, {}, {})\n"
        "  Max Grid Size: ({}, {}, {})\n"
        "  Clock Rate: {} kHz\n"
        "  Compute Capability: {}.{}\n"
        "  Number of streaming multiprocessor: {}\n"
        "  Maximum number of threads: {}\n",
        i,
        prop.name,
        prop.totalGlobalMem,
        prop.sharedMemPerBlock,
        prop.regsPerBlock,
        prop.warpSize,
        prop.maxThreadsPerBlock,
        prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2],
        prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2],
        prop.clockRate / 1000,
        prop.major, prop.minor,
        prop.multiProcessorCount,
        prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount
      );
  }
}

void add()
{
  constexpr int N = 10;
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  for (int i = 0; i < N; ++i)
  {
    a[i] = -i;
    b[i] = i * i;
  }

  CUDA_CHECK(hipMalloc((void **)&dev_a, N * sizeof(int)));
  CUDA_CHECK(hipMalloc((void **)&dev_b, N * sizeof(int)));
  CUDA_CHECK(hipMalloc((void **)&dev_c, N * sizeof(int)));

  CUDA_CHECK(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice));

  kernels::add<<<128, 128>>>(dev_a, dev_b, dev_c, N);

  CUDA_CHECK(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; ++i)
  {
    if (a[i] + b[i] != c[i])
    {
      std::cout << std::format("Error: {} != {} + {}\n", c[i], a[i], b[i]);
    }
  }

  CUDA_CHECK(hipFree(dev_a));
  CUDA_CHECK(hipFree(dev_b));
  CUDA_CHECK(hipFree(dev_c));
}

#define imin(a, b) a<b?a:b
float dot_malloc(){
  constexpr int N = 100*1024*1024;
  constexpr int blocks = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

  float *a, *b, *partial_c;
  a = new float[N];
  b = new float[N];
  partial_c = new float[blocks];

  for(int i = 0; i < N; ++i) {
    a[i] = i;
    b[i] = 2*i;
  }

  float *dev_a, *dev_b, *dev_c;

  hipEvent_t start, stop;

  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipEventRecord(start, 0));

  CUDA_CHECK(hipMalloc((void**)&dev_a, N * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&dev_b, N * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&dev_c, blocks * sizeof(float)));

  CUDA_CHECK(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

  kernels::dot<<<blocks, threadsPerBlock>>>(dev_a, dev_b, dev_c, N);

  CUDA_CHECK(hipMemcpy(partial_c, dev_c, blocks * sizeof(float), hipMemcpyDeviceToHost));

  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));
  float elapsed;
  CUDA_CHECK(hipEventElapsedTime(&elapsed, start, stop));

  float result = 0;
  for(int i = 0; i < blocks; ++i) {
    result += partial_c[i];
  }

  std::cout << std::format("Dot: {}\n", result);
  float _n = N - 1;
  float expected = 2 * (_n * (_n + 1) * (2 * _n + 1) / 6 );

  float diff = std::abs(result - expected);
  float rel_error = diff / std::abs(expected);

  if (rel_error > 1e-5f) {
    std::cout << std::format("Error: {} != {} (rel error = {})\n", result, expected, rel_error);
  }

  CUDA_CHECK(hipFree(dev_a));
  CUDA_CHECK(hipFree(dev_b));
  CUDA_CHECK(hipFree(dev_c));

  delete[] a;
  delete[] b;
  delete[] partial_c;

  return elapsed;
}

float dot_host_malloc(){
  constexpr int N = 100*1024*1024;
  constexpr int blocks = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

  float *a, *b, *partial_c;
  float *dev_a, *dev_b, *dev_c;

  hipEvent_t start, stop;

  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipHostAlloc((void**)&a, N * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
  CUDA_CHECK(hipHostAlloc((void**)&b, N * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
  CUDA_CHECK(hipHostAlloc((void**)&partial_c, blocks * sizeof(float), hipHostMallocMapped));

  CUDA_CHECK(hipHostGetDevicePointer(&dev_a, a, 0));
  CUDA_CHECK(hipHostGetDevicePointer(&dev_b, b, 0));
  CUDA_CHECK(hipHostGetDevicePointer(&dev_c, partial_c, 0));

  for(int i = 0; i < N; ++i) {
    a[i] = i;
    b[i] = 2*i;
  }

  CUDA_CHECK(hipEventRecord(start, 0));

  kernels::dot<<<blocks, threadsPerBlock>>>(dev_a, dev_b, dev_c, N);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));

  float elapsed;
  CUDA_CHECK(hipEventElapsedTime(&elapsed, start, stop));

  float result = 0;
  for(int i = 0; i < blocks; ++i) {
    result += partial_c[i];
  }

  std::cout << std::format("Dot: {}\n", result);
  float _n = N - 1;
  float expected = 2 * (_n * (_n + 1) * (2 * _n + 1) / 6 );

  float diff = std::abs(result - expected);
  float rel_error = diff / std::abs(expected);

  if (rel_error > 1e-5f) {
    std::cout << std::format("Error: {} != {} (rel error = {})\n", result, expected, rel_error);
  }

  CUDA_CHECK(hipHostFree(a));
  CUDA_CHECK(hipHostFree(b));
  CUDA_CHECK(hipHostFree(partial_c));

  return elapsed;
}

void histogram(){
  constexpr int N = 100 * 1024 * 1024;
  std::vector<unsigned char> buffer(N);
  for(int i = 0; i < N; ++i)
  {
    buffer[i] = rand() % 256;
  }

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipEventRecord(start, 0));

  unsigned char* dev_buffer;
  unsigned int* dev_hist;

  CUDA_CHECK(hipMalloc((void**)&dev_buffer, N));
  CUDA_CHECK(hipMemcpy(dev_buffer, buffer.data(), N, hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc((void**)&dev_hist, 256 * sizeof(unsigned int)));
  CUDA_CHECK(hipMemset(dev_hist, 0, 256 * sizeof(unsigned int)));

  std::vector<unsigned int> hist(256);

  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
  int blocks = prop.multiProcessorCount;

  kernels::histogram<<<blocks*2, 256>>>(dev_buffer, N, dev_hist);

  CUDA_CHECK(hipMemcpy(hist.data(), dev_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost));

  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));
  float elapsed_time;
  CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

  std::cout << std::format("Histogram time: {}\n", elapsed_time);

  for(int i = 0; i < N; ++i) {
    hist[buffer[i]]--;
  }
  for(int i = 0; i < 256; ++i) {
    if (hist[i] != 0) {
      std::cout << std::format("Histogram at {} is not zero!\n", i);
    }
  }

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  CUDA_CHECK(hipFree(dev_buffer));
  CUDA_CHECK(hipFree(dev_hist));
}

float cuda_malloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *dev_a;
  float elapsed_time;

  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  std::vector<int> a(size);

  CUDA_CHECK(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));

  CUDA_CHECK(hipEventRecord(start, 0));

  for(int i = 0; i < 100; ++i) {
    if (up) {
      CUDA_CHECK(hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice));
    }
    else {
      CUDA_CHECK(hipMemcpy(a.data(), dev_a, size * sizeof(int), hipMemcpyHostToDevice));
    }
  }

  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
  CUDA_CHECK(hipFree(dev_a));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return elapsed_time;
} 

float cuda_host_malloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsed_time;

  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipHostAlloc((void**)&a, size * sizeof(*a), hipHostMallocDefault));

  CUDA_CHECK(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));

  CUDA_CHECK(hipEventRecord(start, 0));

  for(int i = 0; i < 100; ++i) {
    if (up) {
      CUDA_CHECK(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
    }
    else {
      CUDA_CHECK(hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyHostToDevice));
    }
  }

  CUDA_CHECK(hipHostFree(a));
  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
  CUDA_CHECK(hipFree(dev_a));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return elapsed_time;
}

void memory_time() {
  constexpr int N = 10*1024*1024;
  float elapsed;
  float MB = 100.0f * float(N) * sizeof(int) / 1024.0f / 1024.0f;

  elapsed = cuda_malloc_test(N, true);
  std::cout << std::format("Time using cuda malloc up: {:3.1f} ms\t{:3.1f} MB/s\n", elapsed, MB/(elapsed/1000));
  elapsed = cuda_malloc_test(N, false);
  std::cout << std::format("Time using cuda malloc down: {:3.1f} ms\t{:3.1f} MB/s\n", elapsed, MB/(elapsed/1000));
  elapsed = cuda_host_malloc_test(N, true);
  std::cout << std::format("Time using cuda host malloc up: {:3.1f} ms\t{:3.1f} MB/s\n", elapsed, MB/(elapsed/1000));
  elapsed = cuda_host_malloc_test(N, false);
  std::cout << std::format("Time using cuda host malloc down: {:3.1f} ms\t{:3.1f} MB/s\n", elapsed, MB/(elapsed/1000));
}

void streams() {
  hipDeviceProp_t prop;
  int whichDevice;
  CUDA_CHECK(hipGetDevice(&whichDevice));
  CUDA_CHECK(hipGetDeviceProperties(&prop, whichDevice));
  if (!prop.deviceOverlap) {
    std::cout << "Device will not handle overlaps, so no speedup from streams.\n";
  }

  hipEvent_t start, stop;

  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipEventRecord(start, 0));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;

  int N = 1024*1024;
  int full_data = N*20;

  CUDA_CHECK(hipMalloc((void**)&dev_a, N * sizeof(int)));
  CUDA_CHECK(hipMalloc((void**)&dev_b, N * sizeof(int)));
  CUDA_CHECK(hipMalloc((void**)&dev_c, N * sizeof(int)));

  CUDA_CHECK(hipHostAlloc((void**)&a, full_data * sizeof(int), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&b, full_data * sizeof(int), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&c, full_data * sizeof(int), hipHostMallocDefault));

  for(int i = 0; i < full_data; ++i) {
    a[i] = rand();
    b[i] = rand();
  }

  for(int i = 0; i < full_data; i += N) {
    CUDA_CHECK(hipMemcpyAsync(dev_a, a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(dev_b, b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

    kernels::average<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c, N);

    CUDA_CHECK(hipMemcpyAsync(c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));
  }

  CUDA_CHECK(hipStreamSynchronize(stream));

  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));

  float elapsed;
  CUDA_CHECK(hipEventElapsedTime(&elapsed, start, stop));

  std::cout << std::format("Stream time: {:3.1f} ms\n", elapsed);

  CUDA_CHECK(hipHostFree(a));
  CUDA_CHECK(hipHostFree(b));
  CUDA_CHECK(hipHostFree(c));

  CUDA_CHECK(hipFree(dev_a));
  CUDA_CHECK(hipFree(dev_b));
  CUDA_CHECK(hipFree(dev_c));

  CUDA_CHECK(hipStreamDestroy(stream));
}

void streams_overlapped() {
  hipDeviceProp_t prop;
  int whichDevice;
  CUDA_CHECK(hipGetDevice(&whichDevice));
  CUDA_CHECK(hipGetDeviceProperties(&prop, whichDevice));
  if (!prop.deviceOverlap) {
    std::cout << "Device will not handle overlaps, so no speedup from streams.\n";
  }

  hipEvent_t start, stop;

  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipEventRecord(start, 0));

  hipStream_t stream1, stream2;
  CUDA_CHECK(hipStreamCreate(&stream1));
  CUDA_CHECK(hipStreamCreate(&stream2));

  int *a, *b, *c;
  int *dev_a0, *dev_b0, *dev_c0;
  int *dev_a1, *dev_b1, *dev_c1;

  int N = 1024*1024;
  int full_data = N*20;

  CUDA_CHECK(hipMalloc((void**)&dev_a0, N * sizeof(int)));
  CUDA_CHECK(hipMalloc((void**)&dev_b0, N * sizeof(int)));
  CUDA_CHECK(hipMalloc((void**)&dev_c0, N * sizeof(int)));

  CUDA_CHECK(hipMalloc((void**)&dev_a1, N * sizeof(int)));
  CUDA_CHECK(hipMalloc((void**)&dev_b1, N * sizeof(int)));
  CUDA_CHECK(hipMalloc((void**)&dev_c1, N * sizeof(int)));

  CUDA_CHECK(hipHostAlloc((void**)&a, full_data * sizeof(int), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&b, full_data * sizeof(int), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&c, full_data * sizeof(int), hipHostMallocDefault));

  for(int i = 0; i < full_data; ++i) {
    a[i] = rand();
    b[i] = rand();
  }

  for(int i = 0; i < full_data; i += 2*N) {
    CUDA_CHECK(hipMemcpyAsync(dev_a0, a + i, N * sizeof(int), hipMemcpyHostToDevice, stream1));
    CUDA_CHECK(hipMemcpyAsync(dev_a1, a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream2));

    CUDA_CHECK(hipMemcpyAsync(dev_b0, b + i, N * sizeof(int), hipMemcpyHostToDevice, stream1));
    CUDA_CHECK(hipMemcpyAsync(dev_b1, b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream2));

    kernels::average<<<N/256, 256, 0, stream1>>>(dev_a0, dev_b0, dev_c0, N);
    kernels::average<<<N/256, 256, 0, stream2>>>(dev_a1, dev_b1, dev_c1, N);

    CUDA_CHECK(hipMemcpyAsync(c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
    CUDA_CHECK(hipMemcpyAsync(c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream2));
  }

  CUDA_CHECK(hipStreamSynchronize(stream1));
  CUDA_CHECK(hipStreamSynchronize(stream2));

  CUDA_CHECK(hipEventRecord(stop, 0));

  CUDA_CHECK(hipEventSynchronize(stop));

  float elapsed;
  CUDA_CHECK(hipEventElapsedTime(&elapsed, start, stop));

  std::cout << std::format("Stream overlapped time: {:3.1f} ms\n", elapsed);

  CUDA_CHECK(hipHostFree(a));
  CUDA_CHECK(hipHostFree(b));
  CUDA_CHECK(hipHostFree(c));

  CUDA_CHECK(hipFree(dev_a0));
  CUDA_CHECK(hipFree(dev_b0));
  CUDA_CHECK(hipFree(dev_c0));
  CUDA_CHECK(hipFree(dev_a1));
  CUDA_CHECK(hipFree(dev_b1));
  CUDA_CHECK(hipFree(dev_c1));

  CUDA_CHECK(hipStreamDestroy(stream1));
  CUDA_CHECK(hipStreamDestroy(stream2));
}

void mapped() {
  hipDeviceProp_t prop;
  int whichDevice;
  CUDA_CHECK(hipGetDevice(&whichDevice));
  CUDA_CHECK(hipGetDeviceProperties(&prop, whichDevice));
  if (prop.canMapHostMemory != 1) {
    std::cout << "Device cannot map memory\n";
  }

  CUDA_CHECK(hipSetDeviceFlags(hipDeviceMapHost));
  std::cout << std::format("Regular dot: {:3.1f} ms\nPinned dot: {:3.1f} ms\n", dot_malloc(), dot_host_malloc());
}

int main()
{
  // device_data();
  // add();
  // histogram();
  // memory_time();
  // streams();
  // streams_overlapped();
  // mapped();
  std::cout << std::format("Matrix multiplication time: {} ms\n", cudaviz::matmul(4));
}