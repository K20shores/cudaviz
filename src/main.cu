#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <format>

#include "check_error.hpp"

__global__ void add(int a, int b, int* c) {
  *c = a + b;
}

void deviceData() {
  int deviceCount;

  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  hipDeviceProp_t prop;

  for(int i = 0; i < deviceCount; ++i) {
    CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    std::format("Device {}\n", i);
  }
}

int main () {
  int c;
  int* dev_c;

  CUDA_CHECK(hipMalloc((void**)&dev_c, sizeof(int)));

  add<<<1, 1>>>(2, 3, dev_c);

  CUDA_CHECK(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

  std::cout << "c: " << c << std::endl;

  CUDA_CHECK(hipFree(dev_c));
}